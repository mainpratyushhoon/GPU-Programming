#include "hip/hip_runtime.h"
#include <iostream>
#include<ctime>
using namespace std;
__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}
int random(){
    
    int randomNumber = rand() % 100 + 1;
    return randomNumber;
}
int main() {
    //initialization
    hipEvent_t start,stop;
    srand(time(0));
    int matrixA[3][3];
    int matrixB[3][3];
    int matrixC[3][3]={0};
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //assigning random values to matrix
    for(int i =0;i<3;i++){
        for(int j=0;j<3;j++) matrixA[i][j]=random();
    }
    for(int i =0;i<3;i++){
        for(int j=0;j<3;j++) matrixB[i][j]=random();
    }

    //matrix calculation using cpu and time measurement
    hipEventRecord(start,0);
    for(int i =0;i<3;i++){
        for(int j=0;j<3;j++){
            for(int k=0;k<3;k++){
                matrixC[i][j]+=matrixA[i][k]*matrixB[k][j];
            }
        };
    }
    cout<<endl;
    for(int i =0;i<3;i++){
        for(int j=0;j<3;j++) cout<<matrixC[i][j]<<" ";
        cout<<endl;
    }
    //helloFromGPU<<<1, 1>>>();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float cpu=0;
    hipEventElapsedTime(&cpu,start,stop);
    cout<<cpu;
    hipDeviceSynchronize();
    return 0;
}
