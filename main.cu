#include "hip/hip_runtime.h"
#include <iostream>
#include<ctime>
#define N 1024

using namespace std;
__global__ void multiplyGPU(int *a,int* b,int *c) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if(i<N&&j<N){
        int sum=0;
    for(int k=0;k<N;k++){
        sum+=a[i*N+k]*b[k*N+j];
    }
    c[i*N+j]=sum;
}

}
int random(){
    
    int randomNumber = rand() % 10 + 1;
    return randomNumber;
}
void assignMatrix(int A[N*N]){
    for(int i =0;i<N*N;i++){
        A[i]=random();
    }
}
void multiplyCPU(int matrixA[N*N],int matrixB[N*N],int matrixC[N*N]){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++){
            for(int k=0;k<N;k++){
                matrixC[i*N + j]+=matrixA[i*N+k]*matrixB[k*N+j];
            }
        };
    }
}
void printMatrix(int matrixA[N][N]){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++) cout<<matrixA[i][j]<<" ";
        cout<<endl;
    }
    cout<<endl;
}
void check(int* a,int* b){
    bool correct = true;
    for (int i = 0; i < N * N; i++) {
        if (a[i] != b[i]) {
            correct = false;
            cout << "Mismatch at index " << i << ": CPU=" << a[i] << ", GPU=" << b[i] << endl;
            break;
        }
    }
    if (correct) cout << "CPU and GPU results match!" << endl;
    else cout << "CPU and GPU results do NOT match!" << endl;
}
int main() {
    //initialization
    hipEvent_t start,startG,stop,stopG;
    srand(time(0));

    int *A,*B,*C_cpu,*C_gpu;
    int *d_a,*d_b,*d_c;
    A = (int *)malloc(N * N * sizeof(int));
    B = (int *)malloc(N * N * sizeof(int));
    C_cpu = (int *)malloc(N * N * sizeof(int));
    C_gpu = (int *)malloc(N * N * sizeof(int));

    //assigning values to arrays
    assignMatrix(A);
    assignMatrix(B);
    memset(C_cpu, 0, sizeof(int) * N * N);
    memset(C_gpu, 0, sizeof(int) * N * N);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    //matrix calculation using cpu and time measurement 
    multiplyCPU(A,B,C_cpu);
    cout<<endl;
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float cpu=0;
    hipEventElapsedTime(&cpu,start,stop);

    //matrix calculation using gpu and time measurement
    hipEventCreate(&startG);
    hipEventCreate(&stopG);
    hipEventRecord(startG,0);

    dim3 thread(16,16);
    dim3 blocks((N +15)/16,(N+15)/16);

    if(hipMalloc((void**)&d_a,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of A";
    if(hipMalloc((void**)&d_b,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of B";
    if(hipMalloc((void**)&d_c,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of C";
    if(hipMemcpy(d_a,A,sizeof(int)*N*N,hipMemcpyHostToDevice)!=hipSuccess) cout<<"No copy of A";
    if(hipMemcpy(d_b,B,sizeof(int)*N*N,hipMemcpyHostToDevice)!=hipSuccess) cout<<"No copy of B";

    multiplyGPU<<<blocks, thread>>>(d_a,d_b,d_c);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(C_gpu,d_c,sizeof(int)*N*N,hipMemcpyDeviceToHost);

    hipEventRecord(stopG,0);
    hipEventSynchronize(stopG);
    float gpu=0;
    hipEventElapsedTime(&gpu,startG,stopG);

    cout<<"CPU TIME: "<<cpu<<" ms"<<endl<<"GPU TIME: "<<gpu<<" ms"<<endl;
    hipDeviceSynchronize();

    check(C_cpu,C_gpu);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(A);free(B);free(C_cpu);free(C_gpu);
    return 0;
}
