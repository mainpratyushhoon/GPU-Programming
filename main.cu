#include "hip/hip_runtime.h"
#include <iostream>
#include<ctime>
#define N 289

using namespace std;
__global__ void multiplyGPU(int *a,int* b,int *c) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if(i<N&&j<N){long long sum=0;
    for(int k=0;k<N;k++){
        sum+=a[i*N+k]*b[k*N+j];
    }
    c[i*N+j]=sum;
}

}
int random(){
    
    int randomNumber = rand() % 10 + 1;
    return randomNumber;
}
void assignMatrix(int A[N][N]){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++) A[i][j]=random();
    }
}
void multiplyCPU(int matrixA[N][N],int matrixB[N][N],int matrixC[N][N]){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++){
            for(int k=0;k<N;k++){
                matrixC[i][j]+=matrixA[i][k]*matrixB[k][j];
            }
        };
    }
}
void printMatrix(int matrixA[N][N]){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++) cout<<matrixA[i][j]<<" ";
        cout<<endl;
    }
    cout<<endl;
}
int main() {
    //initialization
    hipEvent_t start,startG,stop,stopG;
    srand(time(0));
    int matrixA[N][N];
    int matrixB[N][N];
    int matrixC[N][N]={0};
    int *d_a,*d_b,*d_c;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    //assigning random values to matrix
    assignMatrix(matrixA);
    //printMatrix(matrixA);
    assignMatrix(matrixB);
    //printMatrix(matrixB);

    //matrix calculation using cpu and time measurement
    hipEventRecord(start,0);
    multiplyCPU(matrixA,matrixB,matrixC);
    cout<<endl;
    //printMatrix(matrixC);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float cpu=0;
    hipEventElapsedTime(&cpu,start,stop);

    hipEventCreate(&startG);
    hipEventCreate(&stopG);
    hipEventRecord(startG,0);
    dim3 thread(16,16);
    dim3 blocks((N +15)/16,(N+15)/16);
    if(hipMalloc((void**)&d_a,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of A";
    if(hipMalloc((void**)&d_b,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of B";
    if(hipMalloc((void**)&d_c,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of C";
    if(hipMemcpy(d_a,matrixA,sizeof(int)*N*N,hipMemcpyHostToDevice)!=hipSuccess) cout<<"No copy of A";
    if(hipMemcpy(d_b,matrixB,sizeof(int)*N*N,hipMemcpyHostToDevice)!=hipSuccess) cout<<"No copy of B";
    multiplyGPU<<<blocks, thread>>>(d_a,d_b,d_c);
    hipError_t err = hipGetLastError();
if (err != hipSuccess) {
    printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
}

    hipMemcpy(matrixC,d_c,sizeof(int)*N*N,hipMemcpyDeviceToHost);
    hipEventRecord(stopG,0);
    hipEventSynchronize(stopG);
    float gpu=0;
    hipEventElapsedTime(&gpu,startG,stopG);
    
    //printMatrix(matrixC);
    cout<<"CPU TIME: "<<cpu<<" ms"<<endl<<"GPU TIME: "<<gpu<<" ms"<<endl;
    hipDeviceSynchronize();
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
