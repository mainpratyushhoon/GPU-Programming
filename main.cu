#include "hip/hip_runtime.h"
#include <iostream>
#include<ctime>
#define N 3
using namespace std;
__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}
int random(){
    
    int randomNumber = rand() % 100 + 1;
    return randomNumber;
}
void assignMatrix(int A[N][N]){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++) A[i][j]=random();
    }
}
void multiplyCPU(int matrixA[N][N],int matrixB[N][N],int matrixC[N][N]){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++){
            for(int k=0;k<N;k++){
                matrixC[i][j]+=matrixA[i][k]*matrixB[k][j];
            }
        };
    }
}
void printMatrix(int matrixA[N][N]){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++) cout<<matrixA[i][j]<<" ";
        cout<<endl;
    }
    cout<<endl;
}
int main() {
    //initialization
    hipEvent_t start,stop;
    srand(time(0));
    int matrixA[N][N];
    int matrixB[N][N];
    int matrixC[N][N]={0};
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //assigning random values to matrix
    assignMatrix(matrixA);
    //printMatrix(matrixA);
    assignMatrix(matrixB);
    //printMatrix(matrixB);
    //matrix calculation using cpu and time measurement
    hipEventRecord(start,0);
    multiplyCPU(matrixA,matrixB,matrixC);
    cout<<endl;
    printMatrix(matrixC);
    //helloFromGPU<<<1, 1>>>();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float cpu=0;
    hipEventElapsedTime(&cpu,start,stop);
    cout<<cpu;
    hipDeviceSynchronize();
    return 0;
}
