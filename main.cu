#include "hip/hip_runtime.h"
#include <iostream>
#include<ctime>
#include<chrono>
using namespace std;
int N;
__global__ void multiplyGPU(int *a,int* b,int *c, int N) {
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if(i<N&&j<N){
        int sum=0;
        for(int k=0;k<N;k++){
            sum+=a[i*N+k]*b[k*N+j];
        }
        c[i*N+j]=sum;
    }
}
int random(){
    int randomNumber = rand() % 10 + 1;
    return randomNumber;
}
void assignMatrix(int* A){
    for(int i =0;i<N*N;i++){
        A[i]=random();
    }
}
void multiplyCPU(int *matrixA,int* matrixB,int* matrixC){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++){
            for(int k=0;k<N;k++){
                matrixC[i*N + j]+=matrixA[i*N+k]*matrixB[k*N+j];
            }
        };
    }
}
void printMatrix(int *matrixA){
    for(int i =0;i<N;i++){
        for(int j=0;j<N;j++) cout<<matrixA[i*N+j]<<" ";
        cout<<endl;
    }
    cout<<endl;
}
void check(int* a,int* b){
    bool correct = true;
    for (int i = 0; i < N * N; i++) {
        if (a[i] != b[i]) {
            correct = false;
            cout << "Mismatch at index " << i << ": CPU=" << a[i] << ", GPU=" << b[i] << endl;
            break;
        }
    }
    if (correct) cout << "CPU and GPU results match!" << endl;
    else cout << "CPU and GPU results do NOT match!" << endl;
}

int main(int argc, char* argv[]){

    if (argc != 2){
        cout << "Usage: ./main <N>" << endl;
        return 1;
    }
    N = atoi(argv[1]);

    //initialization
    hipEvent_t startG,stopG;
    srand(time(0));

    int *A,*B,*C_cpu,*C_gpu;
    int *d_a,*d_b,*d_c;
    A = (int *)malloc(N * N * sizeof(int));
    B = (int *)malloc(N * N * sizeof(int));
    C_cpu = (int *)malloc(N * N * sizeof(int));
    C_gpu = (int *)malloc(N * N * sizeof(int));

    //assigning values to arrays
    assignMatrix(A);
    assignMatrix(B);
    memset(C_cpu, 0, sizeof(int) * N * N);
    memset(C_gpu, 0, sizeof(int) * N * N);

    //matrix calculation using cpu and time measurement 
    auto cpu_start = chrono::high_resolution_clock::now();
    multiplyCPU(A,B,C_cpu);
    cout<<endl;
    auto cpu_end = chrono::high_resolution_clock::now();
    chrono::duration<double> cpu_duration = cpu_end - cpu_start;
    
    //matrix calculation using gpu and time measurement
    hipEventCreate(&startG);
    hipEventCreate(&stopG);
    hipEventRecord(startG,0);

    dim3 thread(2,2);
    dim3 blocks((N +1)/2,(N+1)/2);

    if(hipMalloc((void**)&d_a,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of A";
    if(hipMalloc((void**)&d_b,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of B";
    if(hipMalloc((void**)&d_c,sizeof(int)*N*N)!=hipSuccess) cout<<"No allocation of C";
    if(hipMemcpy(d_a,A,sizeof(int)*N*N,hipMemcpyHostToDevice)!=hipSuccess) cout<<"No copy of A";
    if(hipMemcpy(d_b,B,sizeof(int)*N*N,hipMemcpyHostToDevice)!=hipSuccess) cout<<"No copy of B";

    multiplyGPU<<<blocks, thread>>>(d_a,d_b,d_c,N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(C_gpu,d_c,sizeof(int)*N*N,hipMemcpyDeviceToHost);

    hipEventRecord(stopG,0);
    hipEventSynchronize(stopG);
    float gpu=0;
    hipEventElapsedTime(&gpu,startG,stopG);

    cout << "CPU TIME: " << cpu_duration.count()*1000 << " ms" << endl <<"GPU TIME: "<<gpu<<" ms"<<endl;
    hipDeviceSynchronize();

    check(C_cpu,C_gpu);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(A);free(B);free(C_cpu);free(C_gpu);
    return 0;
}
