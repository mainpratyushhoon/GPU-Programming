
#include <hip/hip_runtime.h>
#include <iostream>
#include<ctime>
using namespace std;
__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}
int main() {
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    helloFromGPU<<<1, 1>>>();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float ms=0;
    hipEventElapsedTime(&ms,start,stop);
    cout<<ms;
    hipDeviceSynchronize();
    return 0;
}
